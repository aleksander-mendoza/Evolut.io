
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int * c){
    *c = a + b;
}

int main(void){
    int c;
    int * dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));
    add<<<1,1>>>(2,7,dev_c);
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("Hello world %d\n", c);
    return 0;
}